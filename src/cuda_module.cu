#include "hip/hip_runtime.h"
#include "cuda_module.h"

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void cuda_add(int* c, const int* a, const int* b, int size) {
    int* d_a, * d_b, * d_c;
    size_t bytes = size * sizeof(int);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_a, d_b, size);
    
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
